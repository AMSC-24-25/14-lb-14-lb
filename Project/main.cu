#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "seconds.h"
#include "LBM.h"
#include <utility>

int main(int argc, char *argv[])
{
    printf("Simulating Lid Driven cavity\n");
    printf("      domain size: %ux%u\n", NX, NY);
    printf("               nu: %g\n", nu);
    printf("              tau: %g\n", tau);
    printf("            u_max: %g\n", u_max);
    printf("             rho0: %g\n", rho0);
    printf("        timesteps: %u\n", NSTEPS);
    printf("       save every: %u\n", NSAVE);
    printf("    message every: %u\n", NMSG);
    printf("\n");

    double bytesPerMiB = 1024.0 * 1024.0;
    double bytesPerGiB = 1024.0 * 1024.0 * 1024.0;

    double *f0;
    double *f1;
    double *f2;
    double *rho;
    double *ux;
    double *uy;

    hipError_t f0_error = hipMallocManaged(&f0, mem_size_0dir);
    hipError_t f1_error = hipMallocManaged(&f1, mem_size_n0dir);
    hipError_t f2_error = hipMallocManaged(&f2, mem_size_n0dir);
    hipError_t rho_error = hipMallocManaged(&rho, mem_size_scalar);
    hipError_t ux_error = hipMallocManaged(&ux, mem_size_scalar);
    hipError_t uy_error = hipMallocManaged(&uy, mem_size_scalar);

    long total_mem_bytes = mem_size_0dir + 2 * mem_size_n0dir + 3 * mem_size_scalar;

    if (
        f0_error != hipSuccess ||
        f1_error != hipSuccess ||
        f2_error != hipSuccess ||
        rho_error != hipSuccess ||
        ux_error != hipSuccess ||
        uy_error != hipSuccess)
    {
        fprintf(stderr, "Error: unable to allocate required memory (%.1f MiB).\n", total_mem_bytes / bytesPerMiB);
        exit(-1);
    }
    else
    {
        printf("Allocated required memory (%.1f MiB).\n", total_mem_bytes / bytesPerMiB);
    }

    // compute lid-driven cavity flow at t=0
    // to initialise rho, ux, uy fields.
    lid_driven_cavity<<<3072, 64>>>(rho, ux, uy);
    hipDeviceSynchronize();
    
    // initialise f1 as equilibrium for rho, ux, uy
    init_equilibrium<<<3072, 64>>>(f0, f1, rho, ux, uy);
    hipDeviceSynchronize();
    
    // Name of CSV
    const char *csv_filename = "simulation_parameters.csv";

    // Write head of CSV
    FILE *csv_file = fopen(csv_filename, "w");
    if (csv_file != nullptr)
    {
        fprintf(csv_file, "NX,NY,NSTEPS,NSAVE,UMAX\n");
        fprintf(csv_file, "%u,%u,%u,%u,%lf\n", NX, NY, NSTEPS, NSAVE, u_max);
        fclose(csv_file);
    }
    else
    {
        fprintf(stderr, "Errore nell'apertura del file CSV %s\n", csv_filename);
    }

    save_scalar("rho", rho, 0);
    save_scalar("ux", ux, 0);
    save_scalar("uy", uy, 0);

    if (computeFlowProperties)
    {
        report_flow_properties(0, rho, ux, uy);
    }

    double start = seconds();

    // main simulation loop; take NSTEPS time steps
    for (unsigned long n = 0; n < NSTEPS; ++n)
    {
        bool save = (n + 1) % NSAVE == 0;
        bool msg = (n + 1) % NMSG == 0;
        bool need_scalars = save || (msg && computeFlowProperties);

        // stream and collide from f1 storing to f2
        // optionally compute and save moments
        stream_collide_save<<<3072, 64>>>(f0, f1, f2, rho, ux, uy, need_scalars);
        hipDeviceSynchronize();
        if (save)
        {

            save_scalar("rho", rho, n + 1);
            save_scalar("ux", ux, n + 1);
            save_scalar("uy", uy, n + 1);
        }

        // swap pointers
        std::swap(f1, f2);

        if (msg)
        {
            if (computeFlowProperties)
            {
                report_flow_properties(n + 1, rho, ux, uy);
            }

            if (!quiet)
                printf("completed timestep %d\n", n + 1);
        }
    }

    double end = seconds();
    double runtime = end - start;

    long doubles_read = ndir; // per node every time step
    long doubles_written = ndir;
    long doubles_saved = 3; // per node every NSAVE time steps

    // note NX*NY overflows when NX=NY=65536
    long nodes_updated = NSTEPS * long(NX * NY);
    long nodes_saved = (NSTEPS / NSAVE) * long(NX * NY);
    double speed = nodes_updated / (1e6 * runtime);

    double bandwidth = (nodes_updated * (doubles_read + doubles_written) + nodes_saved * (doubles_saved)) * sizeof(double) / (runtime * bytesPerGiB);

    printf(" ----- performance information -----\n");
    printf(" memory allocated: %.1f (MiB)\n", total_mem_bytes / bytesPerMiB);
    printf("        timesteps: %u\n", NSTEPS);
    printf("          runtime: %.3f (s)\n", runtime);
    printf("            speed: %.2f (Mlups)\n", speed);
    printf("        bandwidth: %.1f (GiB/s)\n", bandwidth);

    // deallocate memory
    hipFree(f0);
    hipFree(f1);
    hipFree(f2);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);

    return 0;
}